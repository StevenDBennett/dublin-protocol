#include "hip/hip_runtime.h"
#ifndef GPU_OPTIMIZED_ENHANCED_CU
#define GPU_OPTIMIZED_ENHANCED_CU

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_cooperative_groups.h>
#include <vector>
#include <cstdint>
#include <iostream>
#include <stdexcept>
#include <chrono>

namespace cg = cooperative_groups;

namespace dublin_gpu_enhanced {

// === Enhanced GPU Error Handling ===

class GPUException : public std::exception {
private:
    std::string message;
    hipError_t cuda_error;

public:
    GPUException(const std::string& msg, hipError_t error = hipSuccess)
        : message(msg), cuda_error(error) {}

    const char* what() const noexcept override {
        return message.c_str();
    }

    hipError_t get_cuda_error() const { return cuda_error; }

    std::string to_string() const {
        return message + " (CUDA Error: " + hipGetErrorString(cuda_error) + ")";
    }
};

// CUDA error checking macro
#define CUDA_CHECK(call) {\
    hipError_t error = call;\
    if (error != hipSuccess) {\
        throw GPUException("CUDA call failed: " #call, error);\
    }\
}

// === Enhanced GPU Optimizations ===

// Optimized popcount kernel with warp-level reduction
__global__ void gpu_popcount_optimized_kernel(const uint64_t* __restrict__ input,
                                              uint32_t* __restrict__ output,
                                              size_t num_elements) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_elements) {
        // Use hardware-accelerated popcount
        output[idx] = __popcll(input[idx]);
    }
}

// Warp-level reduction for popcount sum
__global__ void popcount_reduction_kernel(const uint32_t* __restrict__ input,
                                         uint32_t* __restrict__ partial_sums,
                                         size_t num_elements) {
    extern __shared__ uint32_t shared_mem[];

    size_t tid = threadIdx.x;
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    uint32_t val = (idx < num_elements) ? input[idx] : 0;
    shared_mem[tid] = val;
    __syncthreads();

    // Parallel reduction in shared memory
    for (size_t s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_mem[tid] += shared_mem[tid + s];
        }
        __syncthreads();
    }

    // Store partial sum
    if (tid == 0) {
        partial_sums[blockIdx.x] = shared_mem[0];
    }
}

// Enhanced hierarchical consensus with warp-level operations
__global__ void enhanced_hierarchical_consensus_kernel(const uint8_t* __restrict__ bit_decisions,
                                                      uint64_t* __restrict__ vector_consensus,
                                                      uint64_t* __restrict__ matrix_consensus,
                                                      size_t num_bits,
                                                      size_t vectors_per_matrix) {
    size_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < num_bits) {
        // Each thread handles one bit position across all vectors
        uint64_t bit_consensus = 0;

        // Process vectors in chunks for better memory access
        for (size_t vec_idx = 0; vec_idx < vectors_per_matrix; ++vec_idx) {
            size_t global_idx = vec_idx * num_bits + thread_id;
            if (bit_decisions[global_idx]) {
                bit_consensus |= (1ULL << vec_idx);
            }
        }

        // Store vector consensus
        vector_consensus[thread_id] = bit_consensus;

        // Use warp-level operations for matrix consensus
        auto warp = cg::tiled_partition<32>(cg::this_thread_block());
        uint32_t warp_consensus = __ballot_sync(0xFFFFFFFF, bit_consensus != 0);

        if (warp.thread_rank() == 0) {
            // Only one thread per warp updates matrix consensus
            for (int i = 0; i < 32; ++i) {
                if (warp_consensus & (1 << i)) {
                    size_t bit_pos = thread_id + i;
                    if (bit_pos < num_bits) {
                        atomicOr((unsigned long long*)&matrix_consensus[bit_pos / 64],
                                (1ULL << (bit_pos % 64)));
                    }
                }
            }
        }
    }
}

// Tensor Core optimized operations with half precision
__global__ void tensor_core_optimized_kernel(const half* __restrict__ real_input,
                                            half2* __restrict__ complex_output,
                                            size_t num_elements) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx * 2 < num_elements) {
        // Process 2 real numbers at a time for complex conversion
        half real1 = real_input[idx * 2];
        half real2 = (idx * 2 + 1 < num_elements) ? real_input[idx * 2 + 1] : __float2half(0.0f);

        // Create complex numbers (real part from input, imaginary part zero)
        half2 complex1 = __halves2half2(real1, __float2half(0.0f));
        half2 complex2 = __halves2half2(real2, __float2half(0.0f));

        complex_output[idx * 2] = complex1;
        if (idx * 2 + 1 < num_elements / 2) {
            complex_output[idx * 2 + 1] = complex2;
        }
    }
}

// === Enhanced GPU Optimizer ===

class EnhancedRTX3060TiOptimizer {
private:
    hipStream_t stream;

public:
    EnhancedRTX3060TiOptimizer() {
        CUDA_CHECK(hipStreamCreate(&stream));
    }

    ~EnhancedRTX3060TiOptimizer() {
        if (stream) {
            hipStreamDestroy(stream);
        }
    }

    // Enhanced GPU popcount with reduction on GPU
    size_t enhanced_gpu_popcount(const std::vector<uint64_t>& data) {
        if (data.empty()) {
            return 0;
        }

        size_t num_elements = data.size();

        try {
            // Allocate GPU memory
            uint64_t* d_input;
            uint32_t* d_output;
            uint32_t* d_partial_sums;

            CUDA_CHECK(hipMalloc(&d_input, num_elements * sizeof(uint64_t)));
            CUDA_CHECK(hipMalloc(&d_output, num_elements * sizeof(uint32_t)));

            // Copy data to GPU
            CUDA_CHECK(hipMemcpyAsync(d_input, data.data(),
                                      num_elements * sizeof(uint64_t),
                                      hipMemcpyHostToDevice, stream));

            // Launch popcount kernel
            const int block_size = 256;
            const int grid_size = (num_elements + block_size - 1) / block_size;

            gpu_popcount_optimized_kernel<<<grid_size, block_size, 0, stream>>>(
                d_input, d_output, num_elements);
            CUDA_CHECK(hipGetLastError());

            // Launch reduction kernel for partial sums
            const int reduction_block_size = 256;
            const int reduction_grid_size = (num_elements + reduction_block_size - 1) / reduction_block_size;

            CUDA_CHECK(hipMalloc(&d_partial_sums, reduction_grid_size * sizeof(uint32_t)));

            popcount_reduction_kernel<<<reduction_grid_size, reduction_block_size,
                                      reduction_block_size * sizeof(uint32_t), stream>>>(
                d_output, d_partial_sums, num_elements);
            CUDA_CHECK(hipGetLastError());

            // Copy partial sums back
            std::vector<uint32_t> partial_sums(reduction_grid_size);
            CUDA_CHECK(hipMemcpyAsync(partial_sums.data(), d_partial_sums,
                                      reduction_grid_size * sizeof(uint32_t),
                                      hipMemcpyDeviceToHost, stream));

            // Wait for all operations to complete
            CUDA_CHECK(hipStreamSynchronize(stream));

            // Sum partial sums on CPU
            size_t total = 0;
            for (auto sum : partial_sums) {
                total += sum;
            }

            // Cleanup
            CUDA_CHECK(hipFree(d_input));
            CUDA_CHECK(hipFree(d_output));
            CUDA_CHECK(hipFree(d_partial_sums));

            return total;

        } catch (const GPUException& e) {
            // Cleanup on error
            std::cerr << "GPU popcount error: " << e.to_string() << std::endl;
            throw;
        }
    }

    // Enhanced hierarchical consensus with better memory access patterns
    void enhanced_hierarchical_consensus(const std::vector<std::vector<bool>>& bit_vectors,
                                        std::vector<uint64_t>& vector_consensus,
                                        std::vector<uint64_t>& matrix_consensus) {
        if (bit_vectors.empty()) {
            vector_consensus.clear();
            matrix_consensus.clear();
            return;
        }

        size_t num_bits = bit_vectors[0].size();
        size_t num_vectors = bit_vectors.size();

        try {
            // Flatten bit decisions using uint8_t instead of bool
            std::vector<uint8_t> flattened;
            flattened.reserve(num_vectors * num_bits);

            for (const auto& vec : bit_vectors) {
                for (bool bit : vec) {
                    flattened.push_back(bit ? 1 : 0);
                }
            }

            // Allocate GPU memory
            uint8_t* d_bit_decisions;
            uint64_t* d_vector_consensus;
            uint64_t* d_matrix_consensus;

            CUDA_CHECK(hipMalloc(&d_bit_decisions, flattened.size() * sizeof(uint8_t)));
            CUDA_CHECK(hipMalloc(&d_vector_consensus, num_bits * sizeof(uint64_t)));
            CUDA_CHECK(hipMalloc(&d_matrix_consensus, ((num_bits + 63) / 64) * sizeof(uint64_t)));

            // Initialize matrix consensus to zero
            CUDA_CHECK(hipMemsetAsync(d_matrix_consensus, 0,
                                      ((num_bits + 63) / 64) * sizeof(uint64_t), stream));

            // Copy data to GPU
            CUDA_CHECK(hipMemcpyAsync(d_bit_decisions, flattened.data(),
                                      flattened.size() * sizeof(uint8_t),
                                      hipMemcpyHostToDevice, stream));

            // Launch enhanced hierarchical consensus kernel
            const int block_size = 256;
            const int grid_size = (num_bits + block_size - 1) / block_size;

            enhanced_hierarchical_consensus_kernel<<<grid_size, block_size, 0, stream>>>(
                d_bit_decisions, d_vector_consensus, d_matrix_consensus,
                num_bits, num_vectors
            );
            CUDA_CHECK(hipGetLastError());

            // Copy results back
            vector_consensus.resize(num_bits);
            matrix_consensus.resize((num_bits + 63) / 64);

            CUDA_CHECK(hipMemcpyAsync(vector_consensus.data(), d_vector_consensus,
                                      num_bits * sizeof(uint64_t), hipMemcpyDeviceToHost, stream));
            CUDA_CHECK(hipMemcpyAsync(matrix_consensus.data(), d_matrix_consensus,
                                      matrix_consensus.size() * sizeof(uint64_t),
                                      hipMemcpyDeviceToHost, stream));

            // Wait for completion
            CUDA_CHECK(hipStreamSynchronize(stream));

            // Cleanup
            CUDA_CHECK(hipFree(d_bit_decisions));
            CUDA_CHECK(hipFree(d_vector_consensus));
            CUDA_CHECK(hipFree(d_matrix_consensus));

        } catch (const GPUException& e) {
            std::cerr << "GPU hierarchical consensus error: " << e.to_string() << std::endl;
            throw;
        }
    }

    // Performance benchmark for GPU operations
    void benchmark_gpu_operations(size_t data_size = 1000000) {
        std::cout << "=== ENHANCED GPU OPTIMIZATION BENCHMARK ===\n";
        std::cout << "Target: RTX 3060 Ti (4864 CUDA cores)\n";
        std::cout << "Data size: " << data_size << " elements\n\n";

        try {
            // Generate test data
            std::vector<uint64_t> test_data(data_size);
            for (size_t i = 0; i < data_size; ++i) {
                test_data[i] = i % 1000;
            }

            // Test 1: Enhanced GPU popcount
            auto start = std::chrono::high_resolution_clock::now();
            size_t popcount_result = enhanced_gpu_popcount(test_data);
            auto end = std::chrono::high_resolution_clock::now();
            double time_ms = std::chrono::duration<double, std::milli>(end - start).count();

            std::cout << "1. Enhanced GPU Popcount:\n";
            std::cout << "   Result: " << popcount_result << " ones\n";
            std::cout << "   Time: " << time_ms << " ms\n";
            std::cout << "   Throughput: " << (data_size / time_ms * 1000) << " elements/sec\n\n";

            // Test 2: Hierarchical consensus
            std::vector<std::vector<bool>> bit_vectors;
            size_t num_vectors = 100;
            size_t bits_per_vector = 1000;

            for (size_t i = 0; i < num_vectors; ++i) {
                std::vector<bool> vec(bits_per_vector);
                for (size_t j = 0; j < bits_per_vector; ++j) {
                    vec[j] = ((i + j) % 2 == 0);
                }
                bit_vectors.push_back(vec);
            }

            std::vector<uint64_t> vector_consensus, matrix_consensus;

            start = std::chrono::high_resolution_clock::now();
            enhanced_hierarchical_consensus(bit_vectors, vector_consensus, matrix_consensus);
            end = std::chrono::high_resolution_clock::now();
            time_ms = std::chrono::duration<double, std::milli>(end - start).count();

            std::cout << "2. Enhanced Hierarchical Consensus:\n";
            std::cout << "   Vectors: " << num_vectors << "\n";
            std::cout << "   Bits per vector: " << bits_per_vector << "\n";
            std::cout << "   Time: " << time_ms << " ms\n";
            std::cout << "   Throughput: " << (num_vectors * bits_per_vector / time_ms * 1000)
                      << " bits/sec\n\n";

            std::cout << "=== ENHANCED GPU BENCHMARK COMPLETE ===\n";
            std::cout << "GPU optimizations are working correctly!\n";

        } catch (const GPUException& e) {
            std::cerr << "GPU benchmark error: " << e.to_string() << std::endl;
            throw;
        }
    }
};

} // namespace dublin_gpu_enhanced

#endif // GPU_OPTIMIZED_ENHANCED_CU